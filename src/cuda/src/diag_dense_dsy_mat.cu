#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>



extern "C" void diag_dn_dsyevd(int n, int *info, double *W, double *A) {

    hipsolverHandle_t cusolverH = NULL;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // Compute eigenvalues and eigenvectors
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER; // Upper triangular part of the matrix is stored

    int lwork = 0;
    double *work = NULL;

    //check_Cusolver_Errors(cusolverDnCreate(&cusolverH), "cusolverDnCreate", __FILE__, __LINE__);
    hipsolverDnCreate(&cusolverH);

    // Query workspace size
    //check_Cusolver_Errors(cusolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, n, A, n, W, &lwork),
    //    "cusolverDnDsyevd_bufferSize", __FILE__, __LINE__);
    //check_Cuda_Errors(cudaMalloc((void**)&work, sizeof(double) * lwork),
    //    "cudaMemcpy", __FILE__, __LINE__);
    hipsolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, n, A, n, W, &lwork);
    hipMalloc((void**)&work, sizeof(double) * lwork);

    // Compute eigenvalues and eigenvectors
    //check_Cusolver_Errors(cusolverDnDsyevd(cusolverH, jobz, uplo, n, A, n, W, work, lwork, info),
    //    "cusolverDnDsyevd", __FILE__, __LINE__);
    hipsolverDnDsyevd(cusolverH, jobz, uplo, n, A, n, W, work, lwork, info);

    // Clean up
    //check_Cuda_Errors(cudaFree(work), "cudaFree", __FILE__, __LINE__);
    //check_Cusolver_Errors(cusolverDnDestroy(cusolverH), "cusolverDnDestroy", __FILE__, __LINE__);

    hipFree(work);
    hipsolverDnDestroy(cusolverH);

}

