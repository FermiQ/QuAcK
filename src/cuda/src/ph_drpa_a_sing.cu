
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ph_dRPA_A_sing_kernel(int nO, int nV, int nBas, int nS, double *eps, double *ERI, double *A) {


    int i, j, a, b;
    int aa, bb;

    long long nVS;
    long long nBas2, nBas3;
    long long i_A0, i_A1, i_A2, i_A3;
    long long i_I0, i_I1, i_I2, i_I3;

    bool a_eq_b;

    nVS = (long long) nV * (long long) nS;

    nBas2 =  (long long) nBas *  (long long) nBas;
    nBas3 = nBas2 *  (long long) nBas;

    aa = blockIdx.x * blockDim.x + threadIdx.x;
    bb = blockIdx.y * blockDim.y + threadIdx.y;

    while(aa < nV) {
        a = aa + nO;

        i_A0 = (long long) aa * (long long) nS;
        i_I0 = (long long) a * nBas2;

        while(bb < nV) {
            b = bb + nO;

            a_eq_b = a == b;

            i_A1 = i_A0 + (long long) bb;
            i_I1 = i_I0 + (long long) b * (long long) nBas;

            i = 0;
            while(i < nO) {

                i_A2 = i_A1 + (long long) i * nVS;
                i_I2 = i_I1 + (long long) i;
 
                j = 0;
                while(j < nO) {

                    i_A3 = i_A2 + (long long) j * (long long) nV;
                    i_I3 = i_I2 + (long long) j * nBas3;

                    A[i_A3] = 2.0 * ERI[i_I3];
                    if(a_eq_b && (i==j)) {
                        A[i_A3] += eps[a] - eps[i];
                    }

                    j ++;
	        } // j

                i ++;
            } // i

            bb += blockDim.y * gridDim.y;
        } // bb

        aa += blockDim.x * gridDim.x;
    } // aa

}





extern "C" void ph_dRPA_A_sing(int nO, int nV, int nBas, int nS, double *eps, double *ERI, double *A) {


    int sBlocks = 32;
    int nBlocks = (nV + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, nBlocks, 1);
    dim3 dimBlock(sBlocks, sBlocks, 1);

    //dim3 dimGrid(nBlocks, 1, 1);
    //dim3 dimBlock(sBlocks, 1, 1);

    printf("lunching ph_dRPA_A_sing_kernel with %dx%d blocks and %dx%d threads/block\n",
        nBlocks, nBlocks, sBlocks, sBlocks);


    ph_dRPA_A_sing_kernel<<<dimGrid, dimBlock>>>(nO, nV, nBas, nS, eps, ERI, A);

}




