
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void A_D_in_B_kernel(int n, double *A, double *D, double *B) {


    int i, j;
    int in, ji;

    double tmp;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;

    while(i < n) {

        in = i * n;

        tmp = D[i];

        while(j < n) {

            ji = in + j;

            B[ji] = A[ji] * tmp;

            j += blockDim.y * gridDim.y;
        } // j

        i += blockDim.x * gridDim.x;
    } // i

}





extern "C" void A_D_in_B(int n, double *A, double *D, double *B) {


    int sBlocks = 32;
    int nBlocks = (n + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, nBlocks, 1);
    dim3 dimBlock(sBlocks, sBlocks, 1);

    printf("lunching A_D_in_B_kernel with %dx%d blocks and %dx%d threads/block\n",
        nBlocks, nBlocks, sBlocks, sBlocks);


    A_D_in_B_kernel<<<dimGrid, dimBlock>>>(n, A, D, B);

}



