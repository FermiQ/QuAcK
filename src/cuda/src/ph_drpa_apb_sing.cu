
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ph_dRPA_ApB_sing_kernel(int nO, int nV, int nBas, int nS, double *eps, double *ERI, double *ApB) {


    int i, j, a, b;
    int aa, bb;
    int nVS;
    int nBas2, nBas3;
    int i_A0, i_A1, i_A2;
    int i_I0, i_I1, i_I2;

    bool a_eq_b;

    nVS = nV * nS;

    nBas2 = nBas * nBas;
    nBas3 = nBas2 * nBas;

    aa = blockIdx.x * blockDim.x + threadIdx.x;
    bb = blockIdx.y * blockDim.y + threadIdx.y;

    while(aa < nV) {
        a = aa + nO;

        i_A0 = aa * nS;
        i_I0 = a * nBas2;

        while(bb < nV) {
            b = bb + nO;

            a_eq_b = a == b;

            i_A1 = i_A0 + bb;
            i_I1 = i_I0 + b * nBas;

            i = 0;
            while(i < nO) {

                i_A2 = i_A1 + i * nVS;
                i_I2 = i_I1 + i;
 
                j = 0;
                while(j < nO) {

                    ApB[i_A2 + j * nV] = 2.0 * (ERI[i_I2 + j * nBas3] + ERI[i_I2 + j * nBas]);
                    if(a_eq_b && (i==j)) {
                        ApB[i_A2 + j * nV] += eps[a] - eps[i];
                    }

                    j ++;
	        } // j

                i ++;
            } // i

            bb += blockDim.y * gridDim.y;
        } // bb

        aa += blockDim.x * gridDim.x;
    } // aa

}





extern "C" void ph_dRPA_ApB_sing(int nO, int nV, int nBas, int nS, double *eps, double *ERI, double *ApB) {


    int sBlocks = 32;
    int nBlocks = (nV + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, nBlocks, 1);
    dim3 dimBlock(sBlocks, sBlocks, 1);

    printf("lunching ph_dRPA_ApB_sing_kernel with %dx%d blocks and %dx%d threads/block\n",
        nBlocks, nBlocks, sBlocks, sBlocks);


    ph_dRPA_ApB_sing_kernel<<<dimGrid, dimBlock>>>(nO, nV, nBas, nS, eps, ERI, ApB);

}




