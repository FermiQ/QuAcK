
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void elementwise_dsqrt_inplace_kernel(int n, double *A) {


    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    while(i < n) {

        if(A[i] > 0.0) {

            A[i] = sqrt(A[i]);

        } else {

            A[i] = sqrt(-A[i]);

        }

        i += blockDim.x * gridDim.x;
    } // i

}





extern "C" void elementwise_dsqrt_inplace(int n, double *A) {

    int sBlocks = 32;
    int nBlocks = (n + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, 1, 1);
    dim3 dimBlock(sBlocks, 1, 1);

    printf("lunching elementwise_dsqrt_inplace_kernel with %d blocks and %d threads/block\n",
        nBlocks, sBlocks);


    elementwise_dsqrt_inplace_kernel<<<dimGrid, dimBlock>>>(n, A);

}




