
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void elementwise_dsqrt_inplace_kernel(int nS, double *A, int *nb_neg_sqrt) {


    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    nb_neg_sqrt = 0;

    while(i < nS) {

        if(A[i] > 0.0) {

            A[i] = sqrt(A[i]);

        } else {

            A[i] = sqrt(-A[i]);

        }

        i += blockDim.x * gridDim.x;
    } // i

}





extern "C" void elementwise_dsqrt_inplace(int nS, double *A, int *nb_neg_sqrt) {

    int sBlocks = 32;
    int nBlocks = (nS + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, 1, 1);
    dim3 dimBlock(sBlocks, 1, 1);

    printf("lunching elementwise_dsqrt_inplace_kernel with %d blocks and %d threads/block\n",
        nBlocks, sBlocks);


    elementwise_dsqrt_inplace_kernel<<<dimGrid, dimBlock>>>(nS, A, nb_neg_sqrt);

}




